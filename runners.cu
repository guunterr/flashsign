#include "hip/hip_runtime.h"
#pragma once

#include "kernels/kernel_1_basic.cuh"
#include "utils.cu"
#include <hipblas.h>

hipblasHandle_t handle;
void run_kernel_basic(int M, int N, int K, bf16 *A, bf16 *B, bf16 *C) {
    const int BK = 8;
    const int TM = 8;
    const int TN = 8;
    const int BM = 64;
    const int BN = 64;
    dim3 gridDim(ceil_div(N, BN), ceil_div(M, BM));
    dim3 blockDim((BM * BN) / (TM * TN));
    kernel6<BM, BN, BK, TM, TN><<<gridDim, blockDim>>>(M, N, K, A, B, C);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error in run_kernel_basic: %s\n", hipGetErrorString(err));
    }
    return;
}

void run_kernel_cublas(int M, int N, int K, bf16 *A, bf16 *B, bf16 *C){

    const float alpha = 1.0f;
    const float beta = 1.0f;
    hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B, HIP_R_16BF, N, A, HIP_R_16BF, K, &beta, C, HIP_R_16BF, N, HIP_R_32F, HIPBLAS_GEMM_DEFAULT);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error in run_kernel_cublas: %s\n", hipGetErrorString(err));
    }
    return;
}

void run_kernel(int kernel_number, int M, int N, int K, bf16 *A, bf16 *B, bf16 *C) {
    switch (kernel_number) {
        case 0:
            run_kernel_cublas(M, N, K, A, B, C);
            break;
        case 1:
            run_kernel_basic(M, N, K, A, B, C);
            break;
        default:
            printf("Invalid kernel number\n");
            break;
    }
    return;
}

void test_kernel(int kernel_number, bool print = false, int N = 256) {
    bf16 *a, *b, *c1, *c2, *d_a, *d_b, *d_c1, *d_c2;
    a = (bf16 *)malloc(N * N * sizeof(bf16));
    b = (bf16 *)malloc(N * N * sizeof(bf16));
    c1 = (bf16 *)malloc(N * N * sizeof(bf16));
    c2 = (bf16 *)malloc(N * N * sizeof(bf16));

    randomise_matrix(a, N * N);
    randomise_matrix(b, N * N);
    randomise_matrix(c1, N * N);
    memcpy(c2, c1, N * N * sizeof(bf16));

    // if (print) {
    //     printf("A: %dx%d\n", N, N);
    //     print_matrix(a, N, N);
    //     printf("B: %dx%d\n", N, N);
    //     print_matrix(b, N, N);
    //     printf("C: %dx%d\n", N, N);
    //     print_matrix(c1, N, N);
    //     printf("C: %dx%d\n", N, N);
    //     print_matrix(c2, N, N);
    // }

    // Allocate memory on device
    hipMalloc((void **)&d_a, N * N * sizeof(bf16));
    hipMalloc((void **)&d_b, N * N * sizeof(bf16));
    hipMalloc((void **)&d_c1, N * N * sizeof(bf16));
    hipMalloc((void **)&d_c2, N * N * sizeof(bf16));

    // Copy data to device
    hipMemcpy(d_a, a, N * N * sizeof(bf16), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * N * sizeof(bf16), hipMemcpyHostToDevice);
    hipMemcpy(d_c1, c1, N * N * sizeof(bf16), hipMemcpyHostToDevice);
    hipMemcpy(d_c2, c2, N * N * sizeof(bf16), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    // Run reference kernel 1 and current kernel
    run_kernel(kernel_number, N, N, N, d_a, d_b, d_c2);
    run_kernel(0, N, N, N, d_a, d_b, d_c1);
    hipDeviceSynchronize();

    // Copy reference kernel 1 and current kernel results back to host
    hipMemcpy(a, d_a, N * N * sizeof(bf16), hipMemcpyDeviceToHost);
    hipMemcpy(b, d_b, N * N * sizeof(bf16), hipMemcpyDeviceToHost);
    hipMemcpy(c2, d_c2, N * N * sizeof(bf16), hipMemcpyDeviceToHost);
    hipMemcpy(c1, d_c1, N * N * sizeof(bf16), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    bool pass = verify_matrix(c2, c1, N * N);
    printf("Kernel %d: %s\n", kernel_number, pass ? "PASS" : "FAIL");
    if (print && !pass) {
        printf("A: %dx%d\n", N, N);
        print_matrix(a, N, N);
        printf("B: %dx%d\n", N, N);
        print_matrix(b, N, N);
        printf("Kernel %d result:\n", kernel_number);
        print_matrix(c1, N, N);
        printf("Reference result:\n");
        print_matrix(c2, N, N);
    }
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c1);
    hipFree(d_c2);
    free(a);
    free(b);
    free(c1);
    free(c2);
    return;
}

void time_kernel(int kernel_number, int N = 1 << 12, int warmup = 2, int runs = 5) {
    bf16 *a, *b, *c, *d_a, *d_b, *d_c;
    printf("Timing kernel %d with %d x %d matrices\n", kernel_number, N, N);
    printf("Warming up...\n");
    for (size_t i = 0; i < warmup; i++) {
        a = make_random_matrix(N, N);
        b = make_random_matrix(N, N);
        c = make_random_matrix(N, N);

        hipMalloc((void **)&d_a, N * N * sizeof(bf16));
        hipMalloc((void **)&d_b, N * N * sizeof(bf16));
        hipMalloc((void **)&d_c, N * N * sizeof(bf16));
        hipMemcpyAsync(d_a, a, N * N * sizeof(bf16), hipMemcpyHostToDevice);
        hipMemcpyAsync(d_b, b, N * N * sizeof(bf16), hipMemcpyHostToDevice);
        hipMemcpyAsync(d_c, c, N * N * sizeof(bf16), hipMemcpyHostToDevice);
        hipDeviceSynchronize();

        run_kernel(kernel_number, N, N, N, d_a, d_b, d_c);

        hipDeviceSynchronize();
    }
    printf("Timing kernel...\n");
    float times[runs];
    for (size_t i = 0; i < runs; i++) {
        // Initialise and copy matrices
        a = make_random_matrix(N, N);
        b = make_random_matrix(N, N);
        c = make_random_matrix(N, N);

        hipMalloc((void **)&d_a, N * N * sizeof(bf16));
        hipMalloc((void **)&d_b, N * N * sizeof(bf16));
        hipMalloc((void **)&d_c, N * N * sizeof(bf16));
        hipMemcpyAsync(d_a, a, N * N * sizeof(bf16), hipMemcpyHostToDevice);
        hipMemcpyAsync(d_b, b, N * N * sizeof(bf16), hipMemcpyHostToDevice);
        hipMemcpyAsync(d_c, c, N * N * sizeof(bf16), hipMemcpyHostToDevice);
        hipDeviceSynchronize();

        // Run and time kernel
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        run_kernel(kernel_number, N, N, N, d_a, d_b, d_c);
        hipEventRecord(stop);
        hipEventSynchronize(start);
        hipEventSynchronize(stop);
        hipDeviceSynchronize();

        hipEventElapsedTime(&times[i], start, stop);
    }
    float average_time = 0;
    float std = 0;
    printf("Kernel %d took the following times:\n", kernel_number);
    for (size_t i = 0; i < runs; i++) {
        printf("%.4f ms\n", times[i]);
        average_time += times[i];
        std += times[i] * times[i];
    }
    average_time /= runs;
    std = sqrt(std / runs - average_time * average_time);
    float relative_error = std / average_time;
    double FLOPS = 2 * pow(N, 3) + pow(N, 2);
    printf("Kernel %d took a total of %.4f+-%.4f ms , doing %.2e FLOPS, giving %.2f +- %.2f TFLOPS/s\n",
           kernel_number, average_time, std, FLOPS, FLOPS / (average_time * 1e9), (FLOPS / (average_time * 1e9)) * relative_error);
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return;
}