#include "hip/hip_runtime.h"
#pragma once

#include "kernels/kernel1.cuh"
#include "kernels/kernel2.cuh"
#include "kernels/kernel3.cuh"
#include "kernels/kernel4.cuh"
#include "kernels/kernel5.cuh"
#include "kernels/kernel6.cuh"
#include "utils.cu"

void run_kernel1(int M, int N, int K, const float *A, const float *B, float *C) {
    dim3 gridDim(ceil_div(M, 32), ceil_div(N, 32));
    dim3 blockDim(32, 32);
    kernel1<<<gridDim, blockDim>>>(M, N, K, A, B, C);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error in run_kernel1: %s\n", hipGetErrorString(err));
    }
    return;
}

void run_kernel2(int M, int N, int K, const float *A, const float *B, float *C) {
    dim3 gridDim(ceil_div(M, 32), ceil_div(N, 32));
    dim3 blockDim(32 * 32);
    kernel2<32><<<gridDim, blockDim>>>(M, N, K, A, B, C);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error in run_kernel2: %s\n", hipGetErrorString(err));
    }
    return;
}

void run_kernel3(int M, int N, int K, const float *A, const float *B, float *C) {
    dim3 gridDim(ceil_div(M, 32), ceil_div(N, 32));
    dim3 blockDim(32 * 32);
    hipFuncSetAttribute(reinterpret_cast<const void*>(kernel3<32>),
                         hipFuncAttributePreferredSharedMemoryCarveout,
                         cudaSharedmemCarveoutMaxShared);
    kernel3<32><<<gridDim, blockDim>>>(M, N, K, A, B, C);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error in run_kernel3: %s\n", hipGetErrorString(err));
    }
    return;
}

void run_kernel4(int M, int N, int K, const float *A, const float *B, float *C) {
    // BK = BN/TM = BM/TM
    const int BK = 8;
    const int TM = 8;
    const int BM = 64;
    const int BN = 64;
    dim3 gridDim(ceil_div(N, BN), ceil_div(M, BM));
    dim3 blockDim((BM * BN) / TM);
    kernel4<BM, BN, BK, TM><<<gridDim, blockDim>>>(M, N, K, A, B, C);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error in run_kernel4: %s\n", hipGetErrorString(err));
    }
    return;
}

void run_kernel5(int M, int N, int K, const float *A, const float *B, float *C) {
    const int BK = 8;
    const int TM = 8;
    const int TN = 8;
    const int BM = 64;
    const int BN = 64;
    dim3 gridDim(ceil_div(N, BN), ceil_div(M, BM));
    dim3 blockDim((BM * BN) / (TM * TN));
    kernel5<BM, BN, BK, TM, TN><<<gridDim, blockDim>>>(M, N, K, A, B, C);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error in run_kernel5: %s\n", hipGetErrorString(err));
    }
    return;
}

void run_kernel6(int M, int N, int K, const float *A, const float *B, float *C) {
    const int BK = 8;
    const int TM = 8;
    const int TN = 8;
    const int BM = 64;
    const int BN = 64;
    dim3 gridDim(ceil_div(N, BN), ceil_div(M, BM));
    dim3 blockDim((BM * BN) / (TM * TN));
    kernel6<BM, BN, BK, TM, TN><<<gridDim, blockDim>>>(M, N, K, A, B, C);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error in run_kernel6: %s\n", hipGetErrorString(err));
    }
    return;
}

void run_kernel(int kernel_number, int M, int N, int K, const float *A, const float *B, float *C) {
    switch (kernel_number) {
        case 1:
            run_kernel1(M, N, K, A, B, C);
            break;
        case 2:
            run_kernel2(M, N, K, A, B, C);
            break;
        case 3:
            run_kernel3(M, N, K, A, B, C);
            break;
        case 4:
            run_kernel4(M, N, K, A, B, C);
            break;
        case 5:
            run_kernel5(M, N, K, A, B, C);
            break;
        case 6:
            run_kernel6(M, N, K, A, B, C);
            break;
        default:
            printf("Invalid kernel number\n");
            break;
    }
    return;
}

void test_kernel(int kernel_number, bool print = false, int N = 256) {
    float *a, *b, *c1, *c2, *d_a, *d_b, *d_c1, *d_c2;
    a = (float *)malloc(N * N * sizeof(float));
    b = (float *)malloc(N * N * sizeof(float));
    c1 = (float *)malloc(N * N * sizeof(float));
    c2 = (float *)malloc(N * N * sizeof(float));

    randomise_matrix(a, N * N);
    randomise_matrix(b, N * N);
    randomise_matrix(c1, N * N);
    memcpy(c2, c1, N * N * sizeof(float));

    // if (print) {
    //     printf("A: %dx%d\n", N, N);
    //     print_matrix(a, N, N);
    //     printf("B: %dx%d\n", N, N);
    //     print_matrix(b, N, N);
    //     printf("C: %dx%d\n", N, N);
    //     print_matrix(c1, N, N);
    //     printf("C: %dx%d\n", N, N);
    //     print_matrix(c2, N, N);
    // }

    // Allocate memory on device
    hipMalloc((void **)&d_a, N * N * sizeof(float));
    hipMalloc((void **)&d_b, N * N * sizeof(float));
    hipMalloc((void **)&d_c1, N * N * sizeof(float));
    hipMalloc((void **)&d_c2, N * N * sizeof(float));

    // Copy data to device
    hipMemcpy(d_a, a, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_c1, c1, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_c2, c2, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    // Run reference kernel 1 and current kernel
    run_kernel(kernel_number, N, N, N, d_a, d_b, d_c2);
    run_kernel(3, N, N, N, d_a, d_b, d_c1);
    hipDeviceSynchronize();

    // Copy reference kernel 1 and current kernel results back to host
    hipMemcpy(a, d_a, N * N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(b, d_b, N * N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(c2, d_c2, N * N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(c1, d_c1, N * N * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    bool pass = verify_matrix(c2, c1, N * N);
    printf("Kernel %d: %s\n", kernel_number, pass ? "PASS" : "FAIL");
    if (print && !pass) {
        printf("A: %dx%d\n", N, N);
        print_matrix(a, N, N);
        printf("B: %dx%d\n", N, N);
        print_matrix(b, N, N);
        printf("Kernel %d result:\n", kernel_number);
        print_matrix(c1, N, N);
        printf("Reference result:\n");
        print_matrix(c2, N, N);
    }
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c1);
    hipFree(d_c2);
    free(a);
    free(b);
    free(c1);
    free(c2);
    return;
}

void time_kernel(int kernel_number, int N = 1 << 12, int warmup = 2, int runs = 5) {
    float *a, *b, *c, *d_a, *d_b, *d_c;
    printf("Timing kernel %d with %d x %d matrices\n", kernel_number, N, N);
    printf("Warming up...\n");
    for (size_t i = 0; i < warmup; i++) {
        a = make_random_matrix(N, N);
        b = make_random_matrix(N, N);
        c = make_random_matrix(N, N);

        hipMalloc((void **)&d_a, N * N * sizeof(float));
        hipMalloc((void **)&d_b, N * N * sizeof(float));
        hipMalloc((void **)&d_c, N * N * sizeof(float));
        hipMemcpyAsync(d_a, a, N * N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpyAsync(d_b, b, N * N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpyAsync(d_c, c, N * N * sizeof(float), hipMemcpyHostToDevice);
        hipDeviceSynchronize();

        run_kernel(kernel_number, N, N, N, d_a, d_b, d_c);

        hipDeviceSynchronize();
    }
    printf("Timing kernel...\n");
    float times[runs];
    for (size_t i = 0; i < runs; i++) {
        // Initialise and copy matrices
        a = make_random_matrix(N, N);
        b = make_random_matrix(N, N);
        c = make_random_matrix(N, N);

        hipMalloc((void **)&d_a, N * N * sizeof(float));
        hipMalloc((void **)&d_b, N * N * sizeof(float));
        hipMalloc((void **)&d_c, N * N * sizeof(float));
        hipMemcpyAsync(d_a, a, N * N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpyAsync(d_b, b, N * N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpyAsync(d_c, c, N * N * sizeof(float), hipMemcpyHostToDevice);
        hipDeviceSynchronize();

        // Run and time kernel
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        run_kernel(kernel_number, N, N, N, d_a, d_b, d_c);
        hipEventRecord(stop);
        hipEventSynchronize(start);
        hipEventSynchronize(stop);
        hipDeviceSynchronize();

        hipEventElapsedTime(&times[i], start, stop);
    }
    float average_time = 0;
    float std = 0;
    printf("Kernel %d took the following times:\n", kernel_number);
    for (size_t i = 0; i < runs; i++) {
        printf("%.2f ms\n", times[i]);
        average_time += times[i];
        std += times[i] * times[i];
    }
    average_time /= runs;
    std = sqrt(std / runs - average_time * average_time);
    double FLOPS = 2 * pow(N, 3) + pow(N, 2);
    printf("Kernel %d took a total of %.2f+-%.2f ms , doing %.2e FLOPS, giving %.2f GFLOPS/s\n",
           kernel_number, average_time, std, FLOPS, FLOPS / (average_time * 1e6));
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return;
}