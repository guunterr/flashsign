#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <cudaTypedefs.h>
#include <hip/hip_fp16.h>
#include <cuda_fp16.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <unistd.h>
#include <mma.h>

#include <cassert>
#include <ctime>
#include <cuda/barrier>
#include <iostream>
#include <random>
#include <vector>

#include "utils.cu"

typedef __half fp16;
typedef __half2 fp162;
namespace test_tensorcore {
#define CUDACHECK(x) { hipError_t err = x; if (err != hipSuccess) { printf("CUDA error in %s: %s\n", __func__, hipGetErrorString(err)); exit(-1); } }
using namespace nvcuda;
namespace cg = cooperative_groups;

int ceil_div(int a, int b) {
    return (a / b) + (a % b != 0);
}

#define flip_buffer(x) (1-x)

#define WARP_SIZE 32

template<const int size>
__device__ void async_load(fp16 *src, fp16 *dst, cg::thread_block block){
    cg::memcpy_async(block, dst, src, size * sizeof(fp16));
}

__device__ void load2(fp16* src, fp16* dst){
    dst[0] = src[0];
    dst[1] = src[1];
}

#define load8_async(src, dst) asm volatile("cp.async.cg.shared.global [%0], [%1], 16;" : : "l"(dst), "l"(src))
#define load2_async(src, dst) asm volatile("cp.async.ca.shared.global [%0], [%1], 4;" : : "l"(dst), "l"(src))

#define load2_async_prefetch(src, dst) asm volatile("cp.async.ca.shared.global.L2::256B [%0], [%1], 4;" : : "l"(dst), "l"(src))
#define load8_async_prefetch(src, dst) asm volatile("cp.async.cg.shared.global.L2::256B [%0], [%1], 16;" : : "l"(dst), "l"(src))

#define cvta_shared_64(addr, smem_ptr) asm volatile("cvta.to.shared.u64 %0, %1;" : "=l"(addr) : "l"(smem_ptr))
#define cvta_shared_32(addr, ptr) asm volatile("cvta.to.shared.u32 %0, %1;" : "=r"(addr) : "r"(ptr))

#define commit_group() asm volatile("cp.async.commit_group;")
#define wait_all() asm volatile("cp.async.wait_all;")

#define ldmatrix1(m, src) asm volatile("ldmatrix.sync.aligned.m8n8.x1.shared.b16 {%0}, [%1];" : "=r"(m) : "l"(src))
#define ldmatrix1_t(m, src) asm volatile("ldmatrix.sync.aligned.m8n8.x1.trans.shared.b16 {%0}, [%1];" : "=r"(m) : "l"(src))

#define ldmatrix2(r0, r1, src) asm volatile("ldmatrix.sync.aligned.m8n8.x2.shared.b16 {%0, %1}, [%2];" : "=r"(r0), "=r"(r1) : "l"(src))

#define ldmatrix4(r0, r1, r2, r3, src) asm volatile(                                    \
    "ldmatrix.sync.aligned.m8n8.x4.shared.b16"                                          \
    "{%0, %1, %2, %3}, [%4];" : "=r"(r0), "=r"(r1), "=r"(r2), "=r"(r3) : "l"(src)       \
)
// #define ldmatrix4(m, src) asm volatile("ldmatrix.sync.")

#define mma_m16n8k8_fp16(a0, a1,b,c0, c1,d0, d1)                            \
    asm volatile(   "{\n\t"                                                 \
                    ".reg .f16x2 %%a<2>, %%b, %%c<2>, %%d<2>; \n\t"         \
                    "mov.b32 %%a0, %5; \n\t"                                \
                    "mov.b32 %%a1, %6; \n\t"                                \
                    "mov.b32 %%b, %2; \n\t"                                 \
                    "mov.b32 %%c0, %3; \n\t"                                \
                    "mov.b32 %%c1, %4; \n\t"                                \
                    "mov.b32 %%d0, %0; \n\t"                                \
                    "mov.b32 %%d1, %1; \n\t"                                \
                    "mma.sync.aligned.m16n8k8.row.col.f16.f16.f16.f16"      \
                    "   {%%d0, %%d1},"                                      \
                    "   {%%a0, %%a1},"                                      \
                    "   {%%b},"                                             \
                    "   {%%c0, %%c1}; \n\t"                                 \
                    "mov.b32 %0, %%d0; \n\t"                                \
                    "mov.b32 %1, %%d1; \n \t"                               \
                    "}"                                                     \
                    : "=r"(d0), "=r"(d1)                                    \
                    : "r"(b), "r"(c0), "r"(c1), "r"(a0), "r"(a1)            \
                )

#define mma_m16n8k16_fp16(a0, a1, a2, a3, b0, b1, c0, c1, d0, d1)           \
    asm volatile(   "{\n\t"                                                 \
                    ".reg .f16x2 %%a<4>, %%b<2>, %%c<2>, %%d<2>; \n\t"      \
                    "mov.b32 %%d0, %0; \n\t"                                \
                    "mov.b32 %%d1, %1; \n\t"                                \
                    "mov.b32 %%a0, %2; \n\t"                                \
                    "mov.b32 %%a1, %3; \n\t"                                \
                    "mov.b32 %%a2, %4; \n\t"                                \
                    "mov.b32 %%a3, %5; \n\t"                                \
                    "mov.b32 %%b0, %6; \n\t"                                \
                    "mov.b32 %%b1, %7; \n\t"                                \
                    "mov.b32 %%c0, %8; \n\t"                                \
                    "mov.b32 %%c1, %9; \n\t"                                \
                    "mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16"     \
                    "   {%%d0, %%d1},"                                      \
                    "   {%%a0, %%a1, %%a2, %%a3},"                          \
                    "   {%%b0, %%b1},"                                      \
                    "   {%%c0, %%c1}; \n\t"                                 \
                    "mov.b32 %0, %%d0; \n\t"                                \
                    "mov.b32 %1, %%d1; \n \t"                               \
                    "}"                                                     \
                    : "=r"(d0), "=r"(d1)                                    \
                    : "r"(a0), "r"(a1), "r"(a2), "r"(a3), "r"(b0), "r"(b1), \
                    "r"(c0), "r"(c1)                                        \
                )

__device__ float2 unpack_half2_cvt_float2(uint32_t packed_floats){
    fp162 floats = reinterpret_cast<fp162*>(&packed_floats)[0];
    return __half22float2(floats);
}

__device__ void preload_kv_chunk();

template<const int BX, const int BY, const int D>
__global__ void kernel(int X, fp16* Q, fp16* K, fp16* V, fp16* O, long long int * global_timers) {
    //Algorithm -> Load Q -> Prefetch KV Row -> Load Next KV Row, Process current dot product and accumualate l2 -> Normalise O -> Output
    const int tix = threadIdx.x;
    const int warpIdx = tix / WARP_SIZE;
    const int laneIdx = tix % WARP_SIZE;
    const int bufferPadding = 8;
    const int D_Frag_Count = D/8;
    __shared__ fp16 KVBuffer[2][2][BX][D + bufferPadding];

    //2 elements = 1 register per 8x8 fragment, D_Frag_Count needed to span D=128, 4 rows required to fill a warp
    //64 registers just for this!
    uint32_t Q_frags[4][D_Frag_Count] = {};
    uint32_t K_frags[2] = {};
    uint32_t S_frags[4][4] = {};
    
    fp16* Q_start = Q + blockIdx.x * BY * D;

    const int access_size = 8;
    const int threads_per_row = D/access_size;
    for (int Q_row = 0; Q_row < 128; Q_row+=access_size)
    {
        fp16* read_ptr = Q_start + Q_row * D + tix*access_size;
        fp16* write_ptr = &KVBuffer[0][0][Q_row + (tix/threads_per_row)][(tix % threads_per_row) * access_size];
        uint64_t write_addr;
        cvta_shared_64(write_addr, write_ptr);
        load8_async_prefetch(read_ptr,write_addr);
    }

    commit_group();
    wait_all();
    __syncthreads();

    //Load Q into registers
    fp16* base_pointer = &KVBuffer[0][0][warpIdx * WARP_SIZE][0];
    for (int D_slice = 0; D_slice < D_Frag_Count; D_slice++)
    {
        fp16* read_ptr = &KVBuffer[0][0][tix][D_slice * 8];
        uint64_t read_addr;
        cvta_shared_64(read_addr, read_ptr);
        ldmatrix4(Q_frags[0][D_slice], Q_frags[1][D_slice], Q_frags[2][D_slice], Q_frags[3][D_slice], read_addr);
    }

    __syncthreads();

    int active_buffer = 0;

    for (int KV_Block = 0; KV_Block < X; KV_Block += BX)
    {
        fp16* K_base_ptr = K + KV_Block * D;
        for (int K_row = 0; K_row < 32; K_row += access_size)
        {
            fp16* read_ptr = K_base_ptr + K_row * D + tix*access_size;
            fp16* write_ptr = &KVBuffer[active_buffer][0][K_row + (tix/threads_per_row)][(tix % threads_per_row) * access_size];
            uint64_t write_addr;
            cvta_shared_64(write_addr, write_ptr);
            load8_async_prefetch(read_ptr,write_addr);
        }
        fp16* V_base_ptr = V + KV_Block * D;
        for (int V_row = 0; V_row < 32; V_row += access_size)
        {
            fp16* read_ptr = V_base_ptr + V_row * D + tix*access_size;
            fp16* write_ptr = &KVBuffer[active_buffer][1][V_row + (tix/threads_per_row)][(tix % threads_per_row) * access_size];
            uint64_t write_addr;
            cvta_shared_64(write_addr, write_ptr);
            load8_async_prefetch(read_ptr,write_addr);
        }
        commit_group();
        wait_all();
        __syncthreads();

        for (int k_chunk = 0; k_chunk < 4; k_chunk++)
        {
            for (int d_chunk = 0; d_chunk < D_Frag_Count; d_chunk+=2)
            {
                fp16* read_ptr = &KVBuffer[active_buffer][0][k_chunk * 8 + laneIdx % 8][8 * (d_chunk + laneIdx / 8)];
                if(tix == 0) printf("Thread 0 reading from block (%d, %d) for k_chunk=%d, d_chunk = %d, %d\n", k_chunk, d_chunk,k_chunk * 8 + laneIdx % 8, 8 * (d_chunk + laneIdx / 8));
                uint64_t read_addr;
                cvta_shared_64(read_addr, read_ptr);
                ldmatrix2(K_frags[0], K_frags[1], read_addr);
                mma_m16n8k16_fp16(Q_frags[0][d_chunk], Q_frags[1][d_chunk], Q_frags[0][d_chunk + 1], Q_frags[1][d_chunk + 1], K_frags[0], K_frags[1], S_frags[0][k_chunk], S_frags[1][k_chunk], S_frags[0][k_chunk], S_frags[1][k_chunk]);
                mma_m16n8k16_fp16(Q_frags[2][d_chunk], Q_frags[3][d_chunk], Q_frags[2][d_chunk + 1], Q_frags[3][d_chunk + 1], K_frags[0], K_frags[1], S_frags[2][k_chunk], S_frags[3][k_chunk], S_frags[2][k_chunk], S_frags[3][k_chunk]);
            }
        }
        __syncthreads();
        
        for (int warp = 0; warp < 4; warp++)
        {
            for (int i = 0; i < 4; i++)
            {
                for (int j = 0; j < 4; j++)
                {
                    for (int thread = 32*warp; thread < 32*(warp + 1); thread++)
                    {
                        if(tix == thread){
                            float2 s2 = unpack_half2_cvt_float2(S_frags[i][j]);
                            int s_row = warp * 32 + i * 8 + laneIdx / 4;
                            int s_col = j * 8 + 2 * (laneIdx % 4);
                            printf("SBLK[%d][%d] in T%d = S[%d][%d] = {%2.2f, %2.2f}\n", i,j,thread, s_row, s_col,s2.x, s2.y);
                        }
                        
                    }
                }
            }
            __syncthreads();
        }

        break;
    }
    
    
}


void run_kernel(int X, int Y, fp16* Q, fp16* K, fp16* V, fp16* O) {
    constexpr uint BY = 128;
    constexpr uint BX = 32;
    constexpr uint D = 128;

    dim3 gridDim(ceil_div(Y, BY));
    dim3 blockDim(BY);
    printf("Running kernel\n");
    kernel<BX, BY, D><<<1, BY>>>(X, Q, K, V, O, 0);
}
}

int main(int argc, char* argv[]) {
    //Allocate and initialise Q, K, V, O
    constexpr int D = 128;
    const int X = 32;
    const int Y = 128;
    fp16 *Q, *K, *V, *O, *d_Q, *d_K, *d_V, *d_O;
    Q = (fp16*)malloc(Y * D * sizeof(fp16));
    K = (fp16*)malloc(X * D * sizeof(fp16));
    V = (fp16*)malloc(X * D * sizeof(fp16));
    O = (fp16*)malloc(Y * D * sizeof(fp16));
    hipMalloc((void**)&d_Q, Y * D * sizeof(fp16));
    hipMalloc((void**)&d_K, X * D * sizeof(fp16));
    hipMalloc((void**)&d_V, X * D * sizeof(fp16));
    hipMalloc((void**)&d_O, Y * D * sizeof(fp16));
    initialise_matrix(Q, Y, D);
    initialise_matrix(K, X, D);
    initialise_matrix(V, X, D);
    hipMemcpy(d_Q, Q, Y * D * sizeof(fp16), hipMemcpyHostToDevice);
    hipMemcpy(d_K, K, X * D * sizeof(fp16), hipMemcpyHostToDevice);
    hipMemcpy(d_V, V, X * D * sizeof(fp16), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    printf("Running kernel\n");
    test_tensorcore::run_kernel(X, Y, d_Q, d_K, d_V, d_O);
    hipDeviceSynchronize();
    hipMemcpy(O, d_O, Y * D * sizeof(fp16), hipMemcpyDeviceToHost);
    return 0;

}
