#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <cudaTypedefs.h>
#include <hip/hip_fp16.h>
#include <cuda_fp16.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <unistd.h>
#include <mma.h>

#include <cassert>
#include <ctime>
#include <cuda/barrier>
#include <iostream>
#include <random>
#include <vector>

namespace flashsign_kernel4 {
#define CUDACHECK(x) { hipError_t err = x; if (err != hipSuccess) { printf("CUDA error in %s: %s\n", __func__, hipGetErrorString(err)); exit(-1); } }
using namespace nvcuda;
namespace cg = cooperative_groups;

typedef __half fp16;
typedef __half2 fp162;

int ceil_div(int a, int b) {
    return (a / b) + (a % b != 0);
}


//64 registers for Q
//64 registers for O
//4 registers for S
//1 register for l
//4 registers for temporary work

template<const int NUM_THREADS, const int size>
__device__ void loadGMEMToSMEM(fp162 *src, fp162 *dst, cg::thread_block& group){
    cg::memcpy_async(group, dst, src, size * sizeof(fp162));
}

template<const int NUM_THREADS, const int size>
__device__ void storeSMEMtoGMEM(fp162 *src, fp162 *dst, cg::thread_block block){
    for (int i = 0; i < size; i+= 4 * NUM_THREADS)
    {
        float4 tmp = reinterpret_cast<float4 *>(&src[i + 4 * threadIdx.x])[0];
        reinterpret_cast<float4 *>(&dst[i + 4 * threadIdx.x])[0] = tmp;
    }
}

__device__ void sync(cg::thread_block& group) {
    cg::wait(group);
}

//64 registers for Q
template<const int BX, const int BY, const int D>
__global__ void kernel(int X, fp16 *Q, fp16 *K, fp16 *V, fp16 *O, long long int *times, bool time = false) {
    cg::thread_block block = cg::this_thread_block();
    //First index is double buffering, second is K vs V
    //[First K buffer][First V Buffer][Second K Buffer][Second V Buffer]
    fp162 __shared__ KVs[2][2][BX * D];
    fp162 regQ[1 * D];
    fp162 regO[1 * D] = {};
    fp162 s2;
    long long int start;
    float l = 0;
    constexpr uint NUM_THREADS = BY;
    //Shuffle Q pointer to the right place
    Q += blockIdx.x * BY * D;
    O += blockIdx.x * BY * D;

    //thread gets its job
    const uint tix = threadIdx.x;

    //thread works with block to load Q
    //Load Q to SMEM in BX Chunks
    //Load those into regQ
    if (tix == 0){
        start = clock64();
    }
    for (uint loadQBXBlock = 0; loadQBXBlock < BY; loadQBXBlock += 2 * BX)
    {   
        //Load a BXxD Chunk of Q into KVs
        loadGMEMToSMEM<NUM_THREADS, 2 * BX * D>(Q, &KVs[0][0][0], block);
        __syncthreads();
        Q += 2 * D * BX;
        //Get that chunk into the appropriate register
        //We're eating some nasty SMEM conflicts here
        int rowIdx = (tix - loadQBXBlock);
        if(rowIdx >= 0 && rowIdx < 2 * BX){
            for (uint i = 0; i < D; i++) regQ[i] = KVs[0][rowIdx / BX][(rowIdx % BX) * D + i];
        }
        __syncthreads();
    }
    if (tix == 0){
        times[blockIdx.x * 8 + 0] = clock64() - start;
        // printf("Q time for block %d: %llu\n", blockIdx.x, times[blockIdx.x * 8 + 0]);
    }
    
    //Prefetch first KV Buffer
    loadGMEMToSMEM<NUM_THREADS, BX * D>(K, &KVs[0][0][0], block);
    loadGMEMToSMEM<NUM_THREADS, BX * D>(V, &KVs[0][1][0], block);
    K += BX * D;
    V += BX * D;
    int active_buffer = 0;
    // Loop over X
    //INVARIANT: Enter an iteration with one memcpy async cooking (on active buffer)
    for (uint KVBlock = 0; KVBlock < X; KVBlock += BX)
    {
        //threads load part of K and V, size BX * D
        loadGMEMToSMEM<NUM_THREADS, BX * D>(K, &KVs[1 - active_buffer][0][0], block);
        loadGMEMToSMEM<NUM_THREADS, BX * D>(V, &KVs[1 - active_buffer][1][0], block);
        //Shuffle K and V forwards
        K += BX * D;
        V += BX * D;
        //Wait for memcpy async from previous iteration
        if (tix == 0){
            start = clock64();
        }
        cg::wait_prior<2>(block);
        if (tix == 0){
            times[blockIdx.x * 8 + 1] += clock64() - start;
            // print("Total KV Wait time for block %d: %llu\n", blockIdx.x, times[blockIdx.x * 8 + 1]);
        }
        if (tix == 0){
            start = clock64();
        }

        //Looping over BX
        for (uint resIdx = 0; resIdx < BX; resIdx+=1)
        {
            //Initialise accumulator to zero
            s2 = __half2half2(HIPRT_ZERO_FP16);

            //Calculate S = QK^T dot product
            for (uint dotIdx = 0; dotIdx < D; dotIdx++)
            {
                
                // fp162 tmp = __hmul2(regQ[dotIdx], KVs[active_buffer][0][resIdx * D + dotIdx]);
                // s2 =__hadd2(s2, tmp);
            }
            //Combine both parts of S (even and odd components of dot on D-axis), duplicate this
            s2 = __half2half2(__hadd(s2.x, s2.y));
            
            //Calculate O = S V
            for (uint dotIdx = 0; dotIdx < D; dotIdx++)
            {
                regO[dotIdx] = __hadd2(regO[dotIdx], __hmul2(s2, KVs[active_buffer][1][resIdx * D + dotIdx]));
            }
            //Calculate l = sum(s^2)
            float s_flt = __half2float(s2.x);
            l += s_flt * s_flt;
        }
        active_buffer = 1 - active_buffer;
        if (tix == 0){
            times[blockIdx.x * 8 + 2] += clock64() - start;
        }
    }
    __syncthreads();
    if (tix == 0){
        start = clock64();
    }
    float rsqrt_l = rsqrt(l);
    fp162 norm_coeff = __float2half2_rn(rsqrt_l);
    for (uint yIdx = 0; yIdx < D; yIdx++)
    {
        regO[yIdx] = __hmul2(regO[yIdx], norm_coeff);
    }
    __syncthreads();
    if (tix == 0){
        times[blockIdx.x * 8 + 3] = clock64() - start;
    }

    if (tix == 0){
        start = clock64();
    }
    for (uint storeOBXBlock = 0; storeOBXBlock < BY; storeOBXBlock += 2 * BX)
    {   
        int rowIdx = (tix - storeOBXBlock);
        if(rowIdx >= 0 && rowIdx < 2 * BX){
            for (int i = 0; i < D; i += 4)
            {
                int write_idx = i;
                reinterpret_cast <float4 *>(&KVs[0][rowIdx / BX][(rowIdx % BX) * D + write_idx])[0] = reinterpret_cast <float4 *>(&regO[write_idx])[0];
            }
        }
        __syncthreads();
        //Load a BXxD Chunk of Q into KVs
        storeSMEMtoGMEM<NUM_THREADS, 2 * BX * D>(&KVs[0][0][0], O, block);
        O += 2 * BX * D;
        __syncthreads();
    }
    if (tix == 0){
        times[blockIdx.x * 8 + 4] = clock64() - start;
    }
    
}

template<const int D>
void run_flashsign4_cuda(int X, int Y, fp16 *Q, fp16 *K, fp16 *V, fp16 *O){
    constexpr int D_HALVED = D / 2;
    constexpr uint BY = 128;
    constexpr uint BX = 8;
    dim3 gridDim(ceil_div(Y, BY));
    dim3 blockDim(BY);
    fp162 *Q_half = reinterpret_cast<fp162 *>(Q);
    fp162 *K_half = reinterpret_cast<fp162 *>(K);
    fp162 *V_half = reinterpret_cast<fp162 *>(V);
    fp162 *O_half = reinterpret_cast<fp162 *>(O);
    // hipFuncSetAttribute(reinterpret_cast<const void*>(kernel<BX), BY, D_HALVED>, hipFuncAttributePreferredSharedMemoryCarveout, 30);
    long long int *times;
    long long int *d_times;
    times = (long long int *)malloc(sizeof(long long int)*8*ceil_div(Y, BY));
    CUDACHECK(hipMalloc((void**)&d_times, sizeof(long long int)*8*ceil_div(Y, BY)));
    hipDeviceSynchronize();
    CUDACHECK(hipMemset(d_times, 0, sizeof(long long int)*8*ceil_div(Y, BY)));
    printf("calling kernel\n");
    hipDeviceSynchronize();
    kernel<BX, BY, D_HALVED><<<gridDim, blockDim>>>(X, Q_half, K_half, V_half, O_half, d_times, true);
    hipDeviceSynchronize();
    printf("Outside kernel\n");
    CUDACHECK(hipGetLastError());
    CUDACHECK(hipMemcpy(times, d_times, sizeof(long long int)*8*ceil_div(Y, BY), hipMemcpyDeviceToHost));
    hipDeviceSynchronize();
    long long int summarised_times[8];
    for (int i = 0; i < ceil_div(Y, BY); i++)
    {
        for (int j = 0; j < 8; j++)
        {
            printf("(%d, %d): %lld ", i, j, times[i * 8 + j]);
            summarised_times[j] += times[i * 8 + j];
        }
        printf("\n");
    }
    printf("Times summarised\n");
    for (int j = 0; j < 8; j++)
    {
        summarised_times[j] /= ceil_div(Y, BY);
    }
    printf("Kernel 4: QLoad = %lld, KVBufferWait = %lld, Dot = %lld, Norm = %lld, Write = %lld\n", summarised_times[0], summarised_times[1], summarised_times[2], summarised_times[3], summarised_times[4]);
    
    hipFree(d_times);
    free(times);
    CUDACHECK(hipGetLastError());
}

template<const int D>
void run_flashsign_4_pytorch(int X, int Y, fp162 *Q, fp162 *K, fp162 *V, fp162 *O){
    constexpr int D_HALVED = D / 2;
    constexpr uint BY = 128;
    constexpr uint BX = 8;
    dim3 gridDim(ceil_div(Y, BY));
    dim3 blockDim(BY);
    kernel<BX, BY, D_HALVED><<<gridDim, blockDim>>>(X, Q, K, V, O, 0);
}
}