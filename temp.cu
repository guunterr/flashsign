#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <cudaTypedefs.h>
#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <unistd.h>

#include <cassert>
#include <ctime>
#include <cuda/barrier>
#include <iostream>
#include <random>
#include <vector>

typedef __hip_bfloat16 bf16;

std::default_random_engine generator = std::default_random_engine(time(0));

void randomise_matrix(bf16 *matrix, int N) {
    std::normal_distribution<float> distribution(0.0, 1.0);
    for (int i = 0; i < N; i++) {
        matrix[i] = __float2bfloat16(distribution(generator));
    }
}

bool verify_matrix(bf16 *matRef, bf16 *matOut, int N) {
    double diff = 0.0;
    int i;
    for (i = 0; i < N; i++) {
        diff = std::fabs(__bfloat162float(matRef[i] - matOut[i]));
        if (diff > 0.1) {
            printf("Divergence! Should %5.2f, Is %5.2f (Diff %5.2f) at %d\n",
                   __bfloat162float(matRef[i]), __bfloat162float(matOut[i]), diff, i);
            return false;
        }
    }
    return true;
}

__global__ void hello() {
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

__global__ void kernel1(int M, int N, int K, const bf16 *A, const bf16 *B, bf16 *C) {
    const uint x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < M && y < N) {
        
        bf16 temp = 0.0;
        for (int i = 0; i < K; ++i) {
            temp += A[x * K + i] * B[i * N + y];
        }
        bf16 new_val = temp + C[x * N + y];
        C[x * N + y] = new_val;
    }
}
template <const uint BLOCKSIZE>
__global__ void kernel2(int M, int N, int K, const bf16 *A, const bf16 *B, bf16 *C) {
    const uint x = blockIdx.x * BLOCKSIZE + (threadIdx.x / BLOCKSIZE);
    const uint y = blockIdx.y * BLOCKSIZE + (threadIdx.x % BLOCKSIZE);

    if (x < M && y < N) {
        bf16 temp = 0.0;
        for (int i = 0; i < K; ++i) {
            temp += A[x * K + i] * B[i * N + y];
        }
        C[x * N + y] = temp + C[x * N + y];
    }
}

int ceil_div(int a, int b) {
    return (a / b) + (a % b != 0);
}

void run_kernel1(int M, int N, int K, const bf16 *A, const bf16 *B, bf16 *C) {
    dim3 gridDim(ceil_div(M, 32), ceil_div(N, 32));
    dim3 blockDim(32, 32);
    kernel1<<<gridDim, blockDim>>>(M, N, K, A, B, C);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error in run_kernel1: %s\n", hipGetErrorString(err));
    }
    return;
}
void run_kernel2(int M, int N, int K, const bf16 *A, const bf16 *B, bf16 *C) {
    dim3 gridDim(ceil_div(M, 32), ceil_div(N, 32));
    dim3 blockDim(32 * 32);
    kernel2<32><<<gridDim, blockDim>>>(M, N, K, A, B, C);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error in run_kernel2: %s\n", hipGetErrorString(err));
    }
    return;
}

void run_kernel(int kernel_number, int M, int N, int K, const bf16 *A, const bf16 *B, bf16 *C) {
    switch (kernel_number) {
        case 1:
            run_kernel1(M, N, K, A, B, C);
            break;
        case 2:
            run_kernel2(M, N, K, A, B, C);
            break;
    }
    return;
}

bf16 *make_random_matrix(int M, int N) {
    bf16 *matrix = (bf16 *)malloc(M * N * sizeof(bf16));
    randomise_matrix(matrix, M * N);
    return matrix;
}

void warmup_kernel() {
    return;
}

void time_kernel(int kernel_number) {
    bf16 *a, *b, *c, *d_a, *d_b, *d_c;
    int N = 1 << 12;
    // Initialise and copy matrices
    a = make_random_matrix(N, N);
    b = make_random_matrix(N, N);
    c = make_random_matrix(N, N);

    hipMalloc((void **)&d_a, N * N * sizeof(bf16));
    hipMalloc((void **)&d_b, N * N * sizeof(bf16));
    hipMalloc((void **)&d_c, N * N * sizeof(bf16));
    hipMemcpyAsync(d_a, a, N * N * sizeof(bf16), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_b, b, N * N * sizeof(bf16), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_c, c, N * N * sizeof(bf16), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    // Run and time kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    run_kernel(kernel_number, N, N, N, d_a, d_b, d_c);
    hipEventRecord(stop);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipDeviceSynchronize();

    // Print time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Kernel %d took %.2f ms, doing %.2f FLOPS, giving %.2f GFLOPS/s\n", kernel_number, milliseconds, (2*pow(N, 3)  + pow(N,2)) , (2*pow(N, 3)  + pow(N,2)) / (milliseconds * 1e6));
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return;
}

void print_matrix(bf16 *matrix, int M, int N) {
    for (size_t i = 0; i < M; i++) {
        for (size_t j = 0; j < N; j++) {
            printf("%6.2f ", __bfloat162float(matrix[i * N + j]));
        }
        printf("\n");
    }
    return;
}

void test_kernel(int kernel_number, bool print = false) {
    bf16 *a, *b, *c1, *c2, *d_a, *d_b, *d_c1, *d_c2;
    int N = 1 << 10;
    a = (bf16 *)malloc(N * N * sizeof(bf16));
    b = (bf16 *)malloc(N * N * sizeof(bf16));
    c1 = (bf16 *)malloc(N * N * sizeof(bf16));
    c2 = (bf16 *)malloc(N * N * sizeof(bf16));

    randomise_matrix(a, N * N);
    randomise_matrix(b, N * N);
    randomise_matrix(c1, N * N);
    memcpy(c2, c1, N * N * sizeof(bf16));

    if (print) {
        printf("A: %dx%d\n", N, N);
        print_matrix(a, N, N);
        printf("B: %dx%d\n", N, N);
        print_matrix(b, N, N);
        printf("C: %dx%d\n", N, N);
        print_matrix(c1, N, N);
        printf("C: %dx%d\n", N, N);
        print_matrix(c2, N, N);
    }

    // Allocate memory on device
    hipMalloc((void **)&d_a, N * N * sizeof(bf16));
    hipMalloc((void **)&d_b, N * N * sizeof(bf16));
    hipMalloc((void **)&d_c1, N * N * sizeof(bf16));
    hipMalloc((void **)&d_c2, N * N * sizeof(bf16));

    // Copy data to device
    hipMemcpy(d_a, a, N * N * sizeof(bf16), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * N * sizeof(bf16), hipMemcpyHostToDevice);
    hipMemcpy(d_c1, c1, N * N * sizeof(bf16), hipMemcpyHostToDevice);
    hipMemcpy(d_c2, c2, N * N * sizeof(bf16), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    // Run reference kernel 1 and current kernel
    run_kernel(kernel_number, N, N, N, d_a, d_b, d_c2);
    run_kernel(1, N, N, N, d_a, d_b, d_c1);
    hipDeviceSynchronize();

    // Copy reference kernel 1 and current kernel results back to host
    hipMemcpy(a, d_a, N * N * sizeof(bf16), hipMemcpyDeviceToHost);
    hipMemcpy(b, d_b, N * N * sizeof(bf16), hipMemcpyDeviceToHost);
    hipMemcpy(c2, d_c2, N * N * sizeof(bf16), hipMemcpyDeviceToHost);
    hipMemcpy(c1, d_c1, N * N * sizeof(bf16), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    bool pass = verify_matrix(c2, c1, N * N);
    if (pass) {
        printf("Kernel %d: %s\n", kernel_number, verify_matrix(c2, c1, N * N) ? "PASS" : "FAIL");
    } else if (print) {
        printf("A: %dx%d\n", N, N);
        print_matrix(a, N, N);
        printf("B: %dx%d\n", N, N);
        print_matrix(b, N, N);
        printf("Kernel %d result:\n", kernel_number);
        print_matrix(c1, N, N);
        printf("Reference result:\n");
        print_matrix(c2, N, N);
    }
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c1);
    hipFree(d_c2);
    free(a);
    free(b);
    free(c1);
    free(c2);
    return;
}

int main(void) {
    // test_kernel(2, false);
    time_kernel(1);
    time_kernel(1);
    time_kernel(1);
    time_kernel(1);

    return 0;
}